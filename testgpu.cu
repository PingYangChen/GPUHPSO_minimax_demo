#include "hip/hip_runtime.h"
#include "testgpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>

// Important Variables
float *hostSwarm1, *hostVeloc1, *hostPBest1, *hostGBest1, *hostSvals1, *hostPvals1, *hostGBval1;
float *hostSwarm2, *hostVeloc2, *hostPBest2, *hostGBest2, *hostSvals2, *hostPvals2, *hostGBval2;
float *GBest1;
float *devSwarm1, *devVeloc1, *devPBest1, *devGBest1, *devSvals1, *devPvals1, *devGBval1;
float *devSwarm2, *devVeloc2, *devPBest2, *devGBest2, *devSvals2, *devPvals2, *devGBval2;

// Global function
__global__ void cudaInitparticles(float *swarm, float *vel, const int loopIdx, unsigned long seed);
__global__ void cudaEvalObjFunc(float *fvals, float *swarm, float *fixed, const int loopIdx);
__global__ void cudaUpdateSwarm(float *swarm, float *vel, float *pBests, float *gBest, const int loopIdx, unsigned long seed);
__global__ void cudaUpdatePBest(float *swarm, float *pBests, float *fvals, float *fpvals, const int loopIdx, const int maximize, const int initial);
__global__ void cudaUpdateGBest(float *pBests, float *gBest, float *fpvals, float *fgval, const int loopIdx, const int maximize, const int initial);

// Device function
__device__ float obj(float x, float y);

//
__device__ float obj(float x, float y) 
{
	float res = (x - 1.0)*(x - 1.0) - y*y;
  return res;
}

//
__global__ void cudaInitparticles(float *swarm, float *vel, const int loopIdx, unsigned long seed) 
{
	int I = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned int seed = i;
	hiprandState state;
	hiprand_init(seed, I, 0, &state);
	
	__shared__ float tmp_swarm[512]; 
	__shared__ float tmp_vel[512];
	
	int LENGTH;
	float U, L; 
	switch (loopIdx) {
		case 0:
			LENGTH = nSwarm1; U = Ux; L = Lx; 
		break;
		case 1:
			LENGTH = nSwarm1 * nSwarm2; U = Uy; L = Ly; 
		break;
	}
	if (i < LENGTH) {
		tmp_swarm[I] = hiprand_uniform(&state) * (U - L) + L;
		swarm[i] = tmp_swarm[I];
		//swarm[i] = hiprand_uniform(&state) * (U - L) + L; 
		//if (i == 0) printf("Loop: %d, S_0: %2.2f\n", loopIdx, swarm[i]);
		tmp_vel[I] = hiprand_uniform(&state);
		vel[i] = tmp_vel[I];
		//vel[i] = hiprand_uniform(&state); 
		//if (i == 0) printf("Loop: %d, V_0: %2.2f\n", loopIdx, vel[i]);
	}	
}

//
__global__ void cudaUpdateSwarm(float *swarm, float *vel, float *pBests, float *gBest, const int loopIdx, unsigned long seed)
{
	int I = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int indexTmp, LENGTH;
	float L, U;
	switch (loopIdx) {
		case 0:
			LENGTH = nSwarm1; 
			indexTmp = (int)(i/nSwarm1);
			L = Lx; U = Ux;
		break;
		case 1:
			LENGTH = nSwarm1 * nSwarm2; 
			indexTmp = (int)(i/nSwarm2);
			L = Ly; U = Uy;
		break;
	}
	
	//unsigned int seed = i;
	hiprandState state;
	hiprand_init(seed, I, 0, &state);
	
	__shared__ float tmp_swarm[512]; 
	__shared__ float tmp_vel[512];
	__shared__ float tmp_pBests[512];
	
	float r1 = hiprand_uniform(&state);
	float r2 = hiprand_uniform(&state);
	
	if (i < LENGTH) {
		//printf("%d: S_iA: %2.2f\n", loopIdx, swarm[i]);
		//if (i == 0) printf("Loop: %d, V_0A: %2.2f\n", loopIdx, vel[i]);
		//printf("%d: pBests: %2.2f\n", loopIdx, pBests[i]);
		//printf("%d: Gindex: %d\n", loopIdx, indexTmp);
		//printf("%d: gBest: %2.2f\n", loopIdx, gBest[indexTmp]);
		tmp_swarm[I] = swarm[i];
		tmp_vel[I] = vel[i];
		tmp_pBests[I] = pBests[i];
		
		tmp_vel[I] = omg * tmp_vel[I] + c1 * r1 * (tmp_pBests[I] - tmp_swarm[I]) + c2 * r2 * (gBest[indexTmp] - tmp_swarm[I]);
		if (tmp_vel[I] > vmax) {
			tmp_vel[I] = vmax;
		}
		if (tmp_vel[I] < -1.0*vmax) {
			tmp_vel[I] = -1.0*vmax;
		}
		tmp_swarm[I] += tmp_vel[I];
		if (tmp_swarm[I] > U) tmp_swarm[I] = U;
		if (tmp_swarm[I] < L) tmp_swarm[I] = L;
		
		swarm[i] = tmp_swarm[I];
		vel[i] = tmp_vel[I];
		/*
		vel[i] = omg * vel[i] + c1 * r1 * (pBests[i] - swarm[i]) + c2 * r2 * (gBest[indexTmp] - swarm[i]);
		if (vel[i] > vmax) {
			vel[i] = vmax;
		}
		if (vel[i] < -1.0*vmax) {
			vel[i] = -1.0*vmax;
		}
		swarm[i] += vel[i];
		if (swarm[i] > U) swarm[i] = U;
		if (swarm[i] < L) swarm[i] = L;
		*/
		//if (i == 0) printf("Loop: %d, V_0B: %2.2f\n", loopIdx, vel[i]);
		//printf("%d: S_i: %2.2f\n", loopIdx, swarm[i]);
	}
}

//
__global__ void cudaUpdatePBest(float *swarm, float *pBests, float *fvals, float *fpvals, const int loopIdx, const int maximize, const int initial) 
{
	int I = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int LENGTH;
	switch (loopIdx) {
		case 0:
			LENGTH = nSwarm1; 
		break;
		case 1:
			LENGTH = nSwarm1 * nSwarm2; 
		break;
	}
	
	__shared__ float tmp_swarm[512]; 
	__shared__ float tmp_pBests[512];
	__shared__ float tmp_fvals[512];
	__shared__ float tmp_fpvals[512];
	
	if (i < LENGTH) {
		tmp_swarm[I] = swarm[i];
		tmp_pBests[I] = pBests[i];
		tmp_fvals[I] = fvals[i];
		tmp_fpvals[I] = fpvals[i];
		
		if (initial == 1) {
			tmp_fpvals[I] = tmp_fvals[I];
			tmp_pBests[I] = tmp_swarm[I];
		} else {
			if (maximize == 1) {
				if (tmp_fvals[I] > tmp_fpvals[I]) {
					tmp_fpvals[I] = tmp_fvals[I];
					tmp_pBests[I] = tmp_swarm[I];
				}
			} else {
				if (tmp_fvals[I] < tmp_fpvals[I]) {
					tmp_fpvals[I] = tmp_fvals[I];
					tmp_pBests[I] = tmp_swarm[I];
				}
			}
		}
		pBests[i] = tmp_pBests[I];
		fpvals[i] = tmp_fpvals[I];
		/*if (initial == 1) {
			fpvals[i] = fvals[i];
			pBests[i] = swarm[i];
		} else {
			if (maximize == 1) {
				if (fvals[i] > fpvals[i]) {
					fpvals[i] = fvals[i];
					pBests[i] = swarm[i];
				}
			} else {
				if (fvals[i] < fpvals[i]) {
					fpvals[i] = fvals[i];
					pBests[i] = swarm[i];
				}
			}
		}*/
	}
}

//
__global__ void cudaUpdateGBest(float *pBests, float *gBest, float *fpvals, float *fgval, const int loopIdx, const int maximize, const int initial)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int lenGBest, lenPBest;
	
	switch (loopIdx) {
		case 0:
			lenGBest = 1; lenPBest = nSwarm1;
		break;
		case 1: 
			lenGBest = nSwarm1; lenPBest = nSwarm2;
		break;
	}
	
	if (i < lenGBest) {
		
		float bestValInEachSwarm = fpvals[i*lenPBest];
		int bestLocInEachSwarm = 0;
		int k;
		if (maximize == 1) {
			for (k = 1; k < lenPBest; k++) {
				if (fpvals[i*lenPBest + k] > bestValInEachSwarm) {
					bestValInEachSwarm = fpvals[i*lenPBest + k];	bestLocInEachSwarm = k;
				}
			}
		} else {
			for (k = 1; k < lenPBest; k++) {
				if (fpvals[i*lenPBest + k] < bestValInEachSwarm) {
					bestValInEachSwarm = fpvals[i*lenPBest + k];	bestLocInEachSwarm = k;
				}
			}
		}
		
		//if (loopIdx == 1) printf("Swarm %d, Gbest: %2.2f \n", i, gBest[i]);
		
		if (initial == 1) {
			fgval[i] = fpvals[i*lenPBest + bestLocInEachSwarm];
			gBest[i] = pBests[i*lenPBest + bestLocInEachSwarm];
		} else {		
			if (maximize == 1) {
				if (fpvals[i*lenPBest + bestLocInEachSwarm] > fgval[i]) {
					fgval[i] = fpvals[i*lenPBest + bestLocInEachSwarm];
					gBest[i] = pBests[i*lenPBest + bestLocInEachSwarm];
				}
			}	else {
				if (fpvals[i*lenPBest + bestLocInEachSwarm] < fgval[i]) {
					fgval[i] = fpvals[i*lenPBest + bestLocInEachSwarm];
					gBest[i] = pBests[i*lenPBest + bestLocInEachSwarm];
				}
			}
		}
		//if (loopIdx == 0) printf("Gbest: %2.2f \n", gBest[i]);
	}
}

__global__ void cudaEvalObjFunc(float *fvals, float *swarm, float *fixed, const int loopIdx)
{
	int I = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int LENGTH;
	int indexTmp;
	
	__shared__ float tmp_swarm[512]; 
	__shared__ float tmp_fvals[512];
		
	switch (loopIdx) {
		case 0:
			LENGTH = nSwarm1; 
			if (i < LENGTH) {
				fvals[i] = fixed[i];				
			}
		break;
		case 1:
			LENGTH = nSwarm1 * nSwarm2; 
			if (i < LENGTH) {
				indexTmp = (int)(i/nSwarm2);
				tmp_swarm[I] = swarm[i];
				tmp_fvals[I] = fvals[i];
				tmp_fvals[I] = obj(fixed[indexTmp], tmp_swarm[I]);
				swarm[i] = tmp_swarm[I];
				fvals[i] = tmp_fvals[I];				
				//fvals[i] = obj(fixed[indexTmp], swarm[i]);
			}			
		break;
	}
}

//
void cudaHPSO()
{
	//
	int threadsNum = 512;
	int blocksNum0 = (int)((nSwarm1 + threadsNum - 1)/threadsNum); 
	int blocksNum1 = (int)(((nSwarm1 * nSwarm2) + threadsNum - 1)/threadsNum); 
	//
	int t1, t2;
	// Start Hierarchical PSO with GPU
	cudaInitparticles<<<blocksNum0, threadsNum>>>(devSwarm1, devVeloc1, 0, unsigned(time(NULL)));
	// --------------------------------------------------------------------------------------------------- //
	cudaInitparticles<<<blocksNum1, threadsNum>>>(devSwarm2, devVeloc2, 1, unsigned(time(NULL))); 
	cudaEvalObjFunc<<<blocksNum1, threadsNum>>>(devSvals2, devSwarm2, devSwarm1, 1); 
	cudaUpdatePBest<<<blocksNum1, threadsNum>>>(devSwarm2, devPBest2, devSvals2, devPvals2, 1, 1, 1);
	cudaUpdateGBest<<<blocksNum0, threadsNum>>>(devPBest2, devGBest2, devPvals2, devGBval2, 1, 1, 1);
	for (t2 = 0; t2 < nIter2; t2++) {
		//printf("2: iteration %d\n", t2);
		cudaUpdateSwarm<<<blocksNum1, threadsNum>>>(devSwarm2, devVeloc2, devPBest2, devGBest2, 1, unsigned(time(NULL)));
		cudaEvalObjFunc<<<blocksNum1, threadsNum>>>(devSvals2, devSwarm2, devSwarm1, 1); 
		cudaUpdatePBest<<<blocksNum1, threadsNum>>>(devSwarm2, devPBest2, devSvals2, devPvals2, 1, 1, 0);
		cudaUpdateGBest<<<blocksNum0, threadsNum>>>(devPBest2, devGBest2, devPvals2, devGBval2, 1, 1, 0);
	}
	// --------------------------------------------------------------------------------------------------- //
	cudaEvalObjFunc<<<blocksNum0, threadsNum>>>(devSvals1, devSwarm1, devGBval2, 0); 
	cudaUpdatePBest<<<blocksNum0, threadsNum>>>(devSwarm1, devPBest1, devSvals1, devPvals1, 0, 0, 1);
	cudaUpdateGBest<<<1, 1>>>(devPBest1, devGBest1, devPvals1, devGBval1, 0, 0, 1);
	for (t1 = 0; t1 < nIter1; t1++) {
		cudaUpdateSwarm<<<blocksNum0, threadsNum>>>(devSwarm1, devVeloc1, devPBest1, devGBest1, 0, unsigned(time(NULL)));
		// --------------------------------------------------------------------------------------------------- //
		cudaInitparticles<<<blocksNum1, threadsNum>>>(devSwarm2, devVeloc2, 1, unsigned(time(NULL))); 
		cudaEvalObjFunc<<<blocksNum1, threadsNum>>>(devSvals2, devSwarm2, devSwarm1, 1); 
		cudaUpdatePBest<<<blocksNum1, threadsNum>>>(devSwarm2, devPBest2, devSvals2, devPvals2, 1, 1, 1);
		cudaUpdateGBest<<<blocksNum0, threadsNum>>>(devPBest2, devGBest2, devPvals2, devGBval2, 1, 1, 1);
		for (t2 = 0; t2 < nIter2; t2++) {
			//printf("2: iteration %d\n", t2);
			cudaUpdateSwarm<<<blocksNum1, threadsNum>>>(devSwarm2, devVeloc2, devPBest2, devGBest2, 1, unsigned(time(NULL)));
			cudaEvalObjFunc<<<blocksNum1, threadsNum>>>(devSvals2, devSwarm2, devSwarm1, 1); 
			cudaUpdatePBest<<<blocksNum1, threadsNum>>>(devSwarm2, devPBest2, devSvals2, devPvals2, 1, 1, 0);
			cudaUpdateGBest<<<blocksNum0, threadsNum>>>(devPBest2, devGBest2, devPvals2, devGBval2, 1, 1, 0);
		}
		// --------------------------------------------------------------------------------------------------- //
		cudaEvalObjFunc<<<blocksNum0, threadsNum>>>(devSvals1, devSwarm1, devGBval2, 0); 
		cudaUpdatePBest<<<blocksNum0, threadsNum>>>(devSwarm1, devPBest1, devSvals1, devPvals1, 0, 0, 0);
		cudaUpdateGBest<<<1, 1>>>(devPBest1, devGBest1, devPvals1, devGBval1, 0, 0, 0);
	}
	// HPSO done
}

//
void Free_Memory() {
	// cleanup
	if (hostSwarm1) free(hostSwarm1);
	if (hostVeloc1) free(hostVeloc1);
	if (hostSvals1) free(hostSvals1);
	if (hostPvals1) free(hostPvals1);
	if (hostGBval1) free(hostGBval1);
	if (hostPBest1) free(hostPBest1);
	if (hostGBest1) free(hostGBest1);
	//
	if (hostSwarm2) free(hostSwarm2);
	if (hostVeloc2) free(hostVeloc2);
	if (hostSvals2) free(hostSvals2);
	if (hostPvals2) free(hostPvals2);
	if (hostGBval2) free(hostGBval2);
	if (hostPBest2) free(hostPBest2);
	if (hostGBest2) free(hostGBest2);
	
	hipError_t fError;	
	if (GBest1) fError = hipHostFree(GBest1); if (fError != 0) printf("CUDA error (free hostGBest1) = %s\n", hipGetErrorString(fError));
	
  if (devSwarm1) fError = hipFree(devSwarm1); if (fError != 0) printf("CUDA error (free devSwarm1) = %s\n", hipGetErrorString(fError));
  if (devVeloc1) fError = hipFree(devVeloc1); if (fError != 0) printf("CUDA error (free devVeloc1) = %s\n", hipGetErrorString(fError));
  if (devSvals1) fError = hipFree(devSvals1); if (fError != 0) printf("CUDA error (free devSvals1) = %s\n", hipGetErrorString(fError));
  if (devPvals1) fError = hipFree(devPvals1); if (fError != 0) printf("CUDA error (free devPvals1) = %s\n", hipGetErrorString(fError));
  if (devGBval1) fError = hipFree(devGBval1); if (fError != 0) printf("CUDA error (free devGBval1) = %s\n", hipGetErrorString(fError));
  if (devPBest1) fError = hipFree(devPBest1); if (fError != 0) printf("CUDA error (free devPBest1) = %s\n", hipGetErrorString(fError));
  if (devGBest1) fError = hipFree(devGBest1); if (fError != 0) printf("CUDA error (free devGBest1) = %s\n", hipGetErrorString(fError));
	//
  if (devSwarm2) fError = hipFree(devSwarm2); if (fError != 0) printf("CUDA error (free devSwarm2) = %s\n", hipGetErrorString(fError));
  if (devVeloc2) fError = hipFree(devVeloc2); if (fError != 0) printf("CUDA error (free devVeloc2) = %s\n", hipGetErrorString(fError));
  if (devSvals2) fError = hipFree(devSvals2); if (fError != 0) printf("CUDA error (free devSvals2) = %s\n", hipGetErrorString(fError));
  if (devPvals2) fError = hipFree(devPvals2); if (fError != 0) printf("CUDA error (free devPvals2) = %s\n", hipGetErrorString(fError));
  if (devGBval2) fError = hipFree(devGBval2); if (fError != 0) printf("CUDA error (free devGBval2) = %s\n", hipGetErrorString(fError));
  if (devPBest2) fError = hipFree(devPBest2); if (fError != 0) printf("CUDA error (free devPBest2) = %s\n", hipGetErrorString(fError));
  if (devGBest2) fError = hipFree(devGBest2); if (fError != 0) printf("CUDA error (free devGBest2) = %s\n", hipGetErrorString(fError));
}

//
void Allocate_Memory() {
	
	size_t alignment = 32;
	posix_memalign((void**)&hostSwarm1, alignment, sizeof(float) * nSwarm1);
	posix_memalign((void**)&hostVeloc1, alignment, sizeof(float) * nSwarm1);
	posix_memalign((void**)&hostSvals1, alignment, sizeof(float) * nSwarm1);
	posix_memalign((void**)&hostPvals1, alignment, sizeof(float) * nSwarm1);
	posix_memalign((void**)&hostGBval1, alignment, sizeof(float) * 1);
	posix_memalign((void**)&hostPBest1, alignment, sizeof(float) * nSwarm1);
	posix_memalign((void**)&hostGBest1, alignment, sizeof(float) * 1);
	
	posix_memalign((void**)&hostSwarm2, alignment, sizeof(float) * nSwarm2);
	posix_memalign((void**)&hostVeloc2, alignment, sizeof(float) * nSwarm2);
	posix_memalign((void**)&hostSvals2, alignment, sizeof(float) * nSwarm2);
	posix_memalign((void**)&hostPvals2, alignment, sizeof(float) * nSwarm2);
	posix_memalign((void**)&hostGBval2, alignment, sizeof(float) * 1);
	posix_memalign((void**)&hostPBest2, alignment, sizeof(float) * nSwarm2);
	posix_memalign((void**)&hostGBest2, alignment, sizeof(float) * 1);
	
	hipError_t Error;
	
  Error = hipHostMalloc((void**)&GBest1, sizeof(float) * 1); 
	if (Error != 0) printf("CUDA error (malloc hostGBest1) = %s\n", hipGetErrorString(Error));
	
  Error = hipMalloc((void**)&devSwarm1, sizeof(float) * nSwarm1); 
	if (Error != 0) printf("CUDA error (malloc devSwarm1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devVeloc1, sizeof(float) * nSwarm1); 
	if (Error != 0) printf("CUDA error (malloc devVeloc1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devSvals1, sizeof(float) * nSwarm1); 
	if (Error != 0) printf("CUDA error (malloc devSvals1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devPvals1, sizeof(float) * nSwarm1); 
	if (Error != 0) printf("CUDA error (malloc devPvals1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devGBval1, sizeof(float) * 1);			 
	if (Error != 0) printf("CUDA error (malloc devGBval1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devPBest1, sizeof(float) * nSwarm1); 
	if (Error != 0) printf("CUDA error (malloc devPBest1) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devGBest1, sizeof(float) * 1);
	if (Error != 0) printf("CUDA error (malloc devGBest1) = %s\n", hipGetErrorString(Error));
	//
  Error = hipMalloc((void**)&devSwarm2, sizeof(float) * nSwarm1 * nSwarm2);
	if (Error != 0) printf("CUDA error (malloc devSwarm2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devVeloc2, sizeof(float) * nSwarm1 * nSwarm2);
	if (Error != 0) printf("CUDA error (malloc devVeloc2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devSvals2, sizeof(float) * nSwarm1 * nSwarm2);
	if (Error != 0) printf("CUDA error (malloc devSvals2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devPvals2, sizeof(float) * nSwarm1 * nSwarm2);
	if (Error != 0) printf("CUDA error (malloc devPvals2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devGBval2, sizeof(float) * nSwarm1);
	if (Error != 0) printf("CUDA error (malloc devGBval2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devPBest2, sizeof(float) * nSwarm1 * nSwarm2);
	if (Error != 0) printf("CUDA error (malloc devPBest2) = %s\n", hipGetErrorString(Error));
  Error = hipMalloc((void**)&devGBest2, sizeof(float) * nSwarm1);
	if (Error != 0) printf("CUDA error (malloc devGBest2) = %s\n", hipGetErrorString(Error));
}

//
void Get_From_Device() {
	hipError_t Error;
	Error = hipMemcpy(GBest1, devGBest1, sizeof(float) * 1, hipMemcpyDeviceToHost);
	if (Error != 0) printf("CUDA error (memcpy devGBest1 -> GBest1) = %s\n", hipGetErrorString(Error));
}